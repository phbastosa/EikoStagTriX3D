#include "hip/hip_runtime.h"
# include "elastic_iso.cuh"

void Elastic_ISO::set_specifications()
{
    modeling_type = "elastic_iso";
    modeling_name = "Modeling type: Elastic isotropic solver";

    auto * Cij = new float[nPoints]();

    std::string vp_file = catch_parameter("vp_model_file", parameters);
    std::string ro_file = catch_parameter("ro_model_file", parameters);
    std::string Cijkl_folder = catch_parameter("Cijkl_folder", parameters);

    float * S = new float[volsize]();
    import_binary_float(vp_file, Cij, nPoints);
    expand_boundary(Cij, S);

    # pragma omp parallel for
    for (int index = 0; index < volsize; index++)
        S[index] = 1.0f / S[index];

    hipMalloc((void**)&(d_S), volsize*sizeof(float));
    hipMemcpy(d_S, S, volsize*sizeof(float), hipMemcpyHostToDevice);
    delete[] S;

    auto * B = new float[volsize]();
    auto * uB = new uintc[volsize]();
    import_binary_float(ro_file, Cij, nPoints);
    expand_boundary(Cij, B);

    # pragma omp parallel for
    for (int index = 0; index < volsize; index++)
        B[index] = 1.0f / B[index];

    compression(B, uB, volsize, maxB, minB);    
    hipMalloc((void**)&(d_B), volsize*sizeof(uintc));
    hipMemcpy(d_B, uB, volsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] B;
    delete[] uB;

    auto * C13 = new float[volsize]();
    auto * uC13 = new uintc[volsize]();
    import_binary_float(Cijkl_folder + "C13.bin", Cij, nPoints);
    expand_boundary(Cij, C13);
    compression(C13, uC13, volsize, maxC13, minC13);    
    hipMalloc((void**)&(d_C13), volsize*sizeof(uintc));
    hipMemcpy(d_C13, uC13, volsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C13;
    delete[] uC13;

    auto * C44 = new float[volsize]();
    auto * uC44 = new uintc[volsize]();
    import_binary_float(Cijkl_folder + "C44.bin", Cij, nPoints);
    expand_boundary(Cij, C44);
    compression(C44, uC44, volsize, maxC44, minC44);    
    hipMalloc((void**)&(d_C44), volsize*sizeof(uintc));
    hipMemcpy(d_C44, uC44, volsize*sizeof(uintc), hipMemcpyHostToDevice);
    delete[] C44;
    delete[] uC44;
}

void Elastic_ISO::compute_eikonal()
{
    float sx = geometry->xsrc[geometry->sInd[srcId]];
    float sy = geometry->ysrc[geometry->sInd[srcId]];
    float sz = geometry->zsrc[geometry->sInd[srcId]];

    time_set<<<nBlocks,nThreads>>>(d_T, volsize);

    dim3 grid(1,1,1);
    dim3 block(MESHDIM,MESHDIM,MESHDIM);

    time_init<<<grid,block>>>(d_T,d_S,sx,sy,sz,dx,dy,dz,sIdx,sIdy,sIdz,nxx,nzz,nb);

    eikonal_solver();
}

void Elastic_ISO::compute_velocity()
{
    compute_velocity_ssg<<<nBlocks,nThreads>>>(d_Vx, d_Vy, d_Vz, d_Txx, d_Tyy, d_Tzz, d_Txz, d_Tyz, d_Txy, d_T, d_B, maxB, minB, d1D, d2D, 
                                               d3D, d_wavelet, dx, dy, dz, dt, timeId, tlag, sIdx, sIdy, sIdz, nxx, nyy, nzz, nb, nt);
}

void Elastic_ISO::compute_pressure()
{
    compute_pressure_ssg<<<nBlocks,nThreads>>>(d_Vx, d_Vy, d_Vz, d_Txx, d_Tyy, d_Tzz, d_Txz, d_Tyz, d_Txy, d_P, d_T, d_C44, d_C13, 
                                               maxC44, minC44, maxC13, minC13, timeId, tlag, dx, dy, dz, dt, nxx, nyy, nzz);    
}

__global__ void compute_velocity_ssg(float * Vx, float * Vy, float * Vz, float * Txx, float * Tyy, float * Tzz, float * Txz, float * Tyz, float * Txy, float * T, uintc * B,
                                     float maxB, float minB, float * damp1D, float * damp2D, float * damp3D, float * wavelet, float dx, float dy, float dz, float dt, int tId, 
                                     int tlag, int sIdx, int sIdy, int sIdz, int nxx, int nyy, int nzz, int nb, int nt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         
    int j = (int) (index - k*nxx*nzz) / nzz;   
    int i = (int) (index - j*nzz - k*nxx*nzz); 

    float Bn, Bm;

    if ((index == 0) && (tId < nt))
    {
        Txx[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
        Tyy[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
        Tzz[sIdz + sIdx*nzz + sIdy*nxx*nzz] += wavelet[tId] / (dx*dy*dz);
    }

    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nyy*nzz))
    {
        Bn = (minB + (static_cast<float>(B[index]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

        if((i >= 3) && (i < nzz-4) && (j > 3) && (j < nxx-3) && (k >= 3) && (k < nyy-4)) 
        {
            float dTxx_dx = (FDM1*(Txx[i + (j-4)*nzz + k*nxx*nzz] - Txx[i + (j+3)*nzz + k*nxx*nzz]) +
                             FDM2*(Txx[i + (j+2)*nzz + k*nxx*nzz] - Txx[i + (j-3)*nzz + k*nxx*nzz]) +
                             FDM3*(Txx[i + (j-2)*nzz + k*nxx*nzz] - Txx[i + (j+1)*nzz + k*nxx*nzz]) +
                             FDM4*(Txx[i + j*nzz + k*nxx*nzz]     - Txx[i + (j-1)*nzz + k*nxx*nzz])) / dx;

            float dTxy_dy = (FDM1*(Txy[i + j*nzz + (k-3)*nxx*nzz] - Txy[i + j*nzz + (k+4)*nxx*nzz]) +
                             FDM2*(Txy[i + j*nzz + (k+3)*nxx*nzz] - Txy[i + j*nzz + (k-2)*nxx*nzz]) +
                             FDM3*(Txy[i + j*nzz + (k-1)*nxx*nzz] - Txy[i + j*nzz + (k+2)*nxx*nzz]) +
                             FDM4*(Txy[i + j*nzz + (k+1)*nxx*nzz] - Txy[i + j*nzz + k*nxx*nzz])) / dy;

            float dTxz_dz = (FDM1*(Txz[(i-3) + j*nzz + k*nxx*nzz] - Txz[(i+4) + j*nzz + k*nxx*nzz]) +
                             FDM2*(Txz[(i+3) + j*nzz + k*nxx*nzz] - Txz[(i-2) + j*nzz + k*nxx*nzz]) +
                             FDM3*(Txz[(i-1) + j*nzz + k*nxx*nzz] - Txz[(i+2) + j*nzz + k*nxx*nzz]) +
                             FDM4*(Txz[(i+1) + j*nzz + k*nxx*nzz] - Txz[i + j*nzz + k*nxx*nzz])) / dz;

            Bm = (minB + (static_cast<float>(B[i + (j+1)*nzz + k*nxx*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

            float Bx = 0.5f*(Bn + Bm);

            Vx[index] += dt*Bx*(dTxx_dx + dTxy_dy + dTxz_dz); 
        }

        if((i >= 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k > 3) && (k < nyy-3)) 
        {
            float dTxy_dx = (FDM1*(Txy[i + (j-3)*nzz + k*nxx*nzz] - Txy[i + (j+4)*nzz + k*nxx*nzz]) +
                             FDM2*(Txy[i + (j+3)*nzz + k*nxx*nzz] - Txy[i + (j-2)*nzz + k*nxx*nzz]) +
                             FDM3*(Txy[i + (j-1)*nzz + k*nxx*nzz] - Txy[i + (j+2)*nzz + k*nxx*nzz]) +
                             FDM4*(Txy[i + (j+1)*nzz + k*nxx*nzz] - Txy[i + j*nzz + k*nxx*nzz])) / dx;

            float dTyy_dy = (FDM1*(Tyy[i + j*nzz + (k-4)*nxx*nzz] - Tyy[i + j*nzz + (k+3)*nxx*nzz]) +
                             FDM2*(Tyy[i + j*nzz + (k+2)*nxx*nzz] - Tyy[i + j*nzz + (k-3)*nxx*nzz]) +
                             FDM3*(Tyy[i + j*nzz + (k-2)*nxx*nzz] - Tyy[i + j*nzz + (k+1)*nxx*nzz]) +
                             FDM4*(Tyy[i + j*nzz + k*nxx*nzz]     - Tyy[i + j*nzz + (k-1)*nxx*nzz])) / dy;

            float dTyz_dz = (FDM1*(Tyz[(i-3) + j*nzz + k*nxx*nzz] - Tyz[(i+4) + j*nzz + k*nxx*nzz]) +
                             FDM2*(Tyz[(i+3) + j*nzz + k*nxx*nzz] - Tyz[(i-2) + j*nzz + k*nxx*nzz]) +
                             FDM3*(Tyz[(i-1) + j*nzz + k*nxx*nzz] - Tyz[(i+2) + j*nzz + k*nxx*nzz]) +
                             FDM4*(Tyz[(i+1) + j*nzz + k*nxx*nzz] - Tyz[i + j*nzz + k*nxx*nzz])) / dz;

            Bm = (minB + (static_cast<float>(B[i + j*nzz + (k+1)*nxx*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

            float By = 0.5f*(Bn + Bm);

            Vy[index] += dt*By*(dTxy_dx + dTyy_dy + dTyz_dz); 
        }    

        if((i > 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k >= 3) && (k < nyy-4)) 
        {
            float dTxz_dx = (FDM1*(Txz[i + (j-3)*nzz + k*nxx*nzz] - Txz[i + (j+4)*nzz + k*nxx*nzz]) +
                             FDM2*(Txz[i + (j+3)*nzz + k*nxx*nzz] - Txz[i + (j-2)*nzz + k*nxx*nzz]) +
                             FDM3*(Txz[i + (j-1)*nzz + k*nxx*nzz] - Txz[i + (j+2)*nzz + k*nxx*nzz]) +
                             FDM4*(Txz[i + (j+1)*nzz + k*nxx*nzz] - Txz[i + j*nzz + k*nxx*nzz])) / dx;

            float dTyz_dy = (FDM1*(Tyz[i + j*nzz + (k-3)*nxx*nzz] - Tyz[i + j*nzz + (k+4)*nxx*nzz]) +
                             FDM2*(Tyz[i + j*nzz + (k+3)*nxx*nzz] - Tyz[i + j*nzz + (k-2)*nxx*nzz]) +
                             FDM3*(Tyz[i + j*nzz + (k-1)*nxx*nzz] - Tyz[i + j*nzz + (k+2)*nxx*nzz]) +
                             FDM4*(Tyz[i + j*nzz + (k+1)*nxx*nzz] - Tyz[i + j*nzz + k*nxx*nzz])) / dy;

            float dTzz_dz = (FDM1*(Tzz[(i-4) + j*nzz + k*nxx*nzz] - Tzz[(i+3) + j*nzz + k*nxx*nzz]) +
                             FDM2*(Tzz[(i+2) + j*nzz + k*nxx*nzz] - Tzz[(i-3) + j*nzz + k*nxx*nzz]) +
                             FDM3*(Tzz[(i-2) + j*nzz + k*nxx*nzz] - Tzz[(i+1) + j*nzz + k*nxx*nzz]) +
                             FDM4*(Tzz[i + j*nzz + k*nxx*nzz]     - Tzz[(i-1) + j*nzz + k*nxx*nzz])) / dz;

            Bm = (minB + (static_cast<float>(B[(i+1) + j*nzz + k*nxx*nzz]) - 1.0f) * (maxB - minB) / (COMPRESS - 1));

            float Bz = 0.5f*(Bn + Bm);

            Vz[index] += dt*Bz*(dTxz_dx + dTyz_dy + dTzz_dz); 
        }

    	float damper = get_boundary_damper(damp1D, damp2D, damp3D, i, j, k, nxx, nyy, nzz, nb);

        Vx[index] *= damper;
        Vy[index] *= damper;
        Vz[index] *= damper;

        Txx[index] *= damper;
        Tyy[index] *= damper;
        Tzz[index] *= damper;
        Txz[index] *= damper;
        Tyz[index] *= damper;
        Txy[index] *= damper;
    }
}

__global__ void compute_pressure_ssg(float * Vx, float * Vy, float * Vz, float * Txx, float * Tyy, float * Tzz, float * Txz, float * Tyz, float * Txy, float * P, float * T, uintc * C44, 
                                     uintc * C13, float maxC44, float minC44, float maxC13, float minC13, int tId, int tlag, float dx, float dy, float dz, float dt, int nxx, int nyy, int nzz)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         
    int j = (int) (index - k*nxx*nzz) / nzz;   
    int i = (int) (index - j*nzz - k*nxx*nzz); 

    float c44_1, c44_2, c44_3, c44_4;

    if ((T[index] < (float)(tId + tlag)*dt) && (index < nxx*nyy*nzz))
    {
        if((i >= 3) && (i < nzz-4) && (j >= 3) && (j < nxx-4) && (k >= 3) && (k < nyy-4)) 
        {    
            float dVx_dx = (FDM1*(Vx[i + (j-3)*nzz + k*nxx*nzz] - Vx[i + (j+4)*nzz + k*nxx*nzz]) +
                            FDM2*(Vx[i + (j+3)*nzz + k*nxx*nzz] - Vx[i + (j-2)*nzz + k*nxx*nzz]) +
                            FDM3*(Vx[i + (j-1)*nzz + k*nxx*nzz] - Vx[i + (j+2)*nzz + k*nxx*nzz]) +
                            FDM4*(Vx[i + (j+1)*nzz + k*nxx*nzz] - Vx[i + j*nzz + k*nxx*nzz])) / dx;

            float dVy_dy = (FDM1*(Vy[i + j*nzz + (k-3)*nxx*nzz] - Vy[i + j*nzz + (k+4)*nxx*nzz]) +
                            FDM2*(Vy[i + j*nzz + (k+3)*nxx*nzz] - Vy[i + j*nzz + (k-2)*nxx*nzz]) +
                            FDM3*(Vy[i + j*nzz + (k-1)*nxx*nzz] - Vy[i + j*nzz + (k+2)*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + (k+1)*nxx*nzz] - Vy[i + j*nzz + k*nxx*nzz])) / dy;

            float dVz_dz = (FDM1*(Vz[(i-3) + j*nzz + k*nxx*nzz] - Vz[(i+4) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vz[(i+3) + j*nzz + k*nxx*nzz] - Vz[(i-2) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vz[(i-1) + j*nzz + k*nxx*nzz] - Vz[(i+2) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vz[(i+1) + j*nzz + k*nxx*nzz] - Vz[i + j*nzz + k*nxx*nzz])) / dz;
            
            float c13 = (minC13 + (static_cast<float>(C13[index]) - 1.0f) * (maxC13 - minC13) / (COMPRESS - 1));
            float c44 = (minC44 + (static_cast<float>(C44[index]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));

            Txx[index] += dt*((c13 + 2*c44)*dVx_dx + c13*(dVy_dy + dVz_dz));
            Tyy[index] += dt*((c13 + 2*c44)*dVy_dy + c13*(dVx_dx + dVz_dz));
            Tzz[index] += dt*((c13 + 2*c44)*dVz_dz + c13*(dVx_dx + dVy_dy));                    
        }

        if((i >= 3) && (i < nzz-4) && (j > 3) && (j < nxx-3) && (k > 3) && (k < nyy-3)) 
        {
            float dVx_dy = (FDM1*(Vx[i + j*nzz + (k-4)*nxx*nzz] - Vx[i + j*nzz + (k+3)*nxx*nzz]) +
                            FDM2*(Vx[i + j*nzz + (k+2)*nxx*nzz] - Vx[i + j*nzz + (k-3)*nxx*nzz]) +
                            FDM3*(Vx[i + j*nzz + (k-2)*nxx*nzz] - Vx[i + j*nzz + (k+1)*nxx*nzz]) +
                            FDM4*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[i + j*nzz + (k-1)*nxx*nzz])) / dy;

            float dVy_dx = (FDM1*(Vy[i + (j-4)*nzz + k*nxx*nzz] - Vy[i + (j+3)*nzz + k*nxx*nzz]) +
                            FDM2*(Vy[i + (j+2)*nzz + k*nxx*nzz] - Vy[i + (j-3)*nzz + k*nxx*nzz]) +
                            FDM3*(Vy[i + (j-2)*nzz + k*nxx*nzz] - Vy[i + (j+1)*nzz + k*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[i + (j-1)*nzz + k*nxx*nzz])) / dx;

            c44_1 = (minC44 + (static_cast<float>(C44[i + (j+1)*nzz + (k+1)*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_2 = (minC44 + (static_cast<float>(C44[i + (j+1)*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_3 = (minC44 + (static_cast<float>(C44[i + j*nzz + (k+1)*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_4 = (minC44 + (static_cast<float>(C44[i + j*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));

            float Mxy = powf(0.25f*(1.0f/c44_1 + 1.0f/c44_2 + 1.0f/c44_3 + 1.0f/c44_4),-1.0f);

            Txy[index] += dt*Mxy*(dVx_dy + dVy_dx);
        }

        if((i > 3) && (i < nzz-3) && (j > 3) && (j < nxx-3) && (k >= 3) && (k < nyy-4)) 
        {
            float dVx_dz = (FDM1*(Vx[(i-4) + j*nzz + k*nxx*nzz] - Vx[(i+3) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vx[(i+2) + j*nzz + k*nxx*nzz] - Vx[(i-3) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vx[(i-2) + j*nzz + k*nxx*nzz] - Vx[(i+1) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vx[i + j*nzz + k*nxx*nzz]     - Vx[(i-1) + j*nzz + k*nxx*nzz])) / dz;

            float dVz_dx = (FDM1*(Vz[i + (j-4)*nzz + k*nxx*nzz] - Vz[i + (j+3)*nzz + k*nxx*nzz]) +
                            FDM2*(Vz[i + (j+2)*nzz + k*nxx*nzz] - Vz[i + (j-3)*nzz + k*nxx*nzz]) +
                            FDM3*(Vz[i + (j-2)*nzz + k*nxx*nzz] - Vz[i + (j+1)*nzz + k*nxx*nzz]) +
                            FDM4*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + (j-1)*nzz + k*nxx*nzz])) / dx;

            c44_1 = (minC44 + (static_cast<float>(C44[(i+1) + (j+1)*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_2 = (minC44 + (static_cast<float>(C44[i + (j+1)*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_3 = (minC44 + (static_cast<float>(C44[(i+1) + j*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_4 = (minC44 + (static_cast<float>(C44[i + j*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));

            float Mxz = powf(0.25f*(1.0f/c44_1 + 1.0f/c44_2 + 1.0f/c44_3 + 1.0f/c44_4),-1.0f);

            Txz[index] += dt*Mxz*(dVx_dz + dVz_dx);
        }

        if((i > 3) && (i < nzz-3) && (j >= 3) && (j < nxx-4) && (k > 3) && (k < nyy-3)) 
        {
            float dVy_dz = (FDM1*(Vy[(i-4) + j*nzz + k*nxx*nzz] - Vy[(i+3) + j*nzz + k*nxx*nzz]) +
                            FDM2*(Vy[(i+2) + j*nzz + k*nxx*nzz] - Vy[(i-3) + j*nzz + k*nxx*nzz]) +
                            FDM3*(Vy[(i-2) + j*nzz + k*nxx*nzz] - Vy[(i+1) + j*nzz + k*nxx*nzz]) +
                            FDM4*(Vy[i + j*nzz + k*nxx*nzz]     - Vy[(i-1) + j*nzz + k*nxx*nzz])) / dz;

            float dVz_dy = (FDM1*(Vz[i + j*nzz + (k-4)*nxx*nzz] - Vz[i + j*nzz + (k+3)*nxx*nzz]) +
                            FDM2*(Vz[i + j*nzz + (k+2)*nxx*nzz] - Vz[i + j*nzz + (k-3)*nxx*nzz]) +
                            FDM3*(Vz[i + j*nzz + (k-2)*nxx*nzz] - Vz[i + j*nzz + (k+1)*nxx*nzz]) +
                            FDM4*(Vz[i + j*nzz + k*nxx*nzz]     - Vz[i + j*nzz + (k-1)*nxx*nzz])) / dy;
            
            c44_1 = (minC44 + (static_cast<float>(C44[(i+1) + j*nzz + (k+1)*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_2 = (minC44 + (static_cast<float>(C44[i + j*nzz + (k+1)*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_3 = (minC44 + (static_cast<float>(C44[(i+1) + j*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));
            c44_4 = (minC44 + (static_cast<float>(C44[i + j*nzz + k*nxx*nzz]) - 1.0f) * (maxC44 - minC44) / (COMPRESS - 1));

            float Myz = powf(0.25f*(1.0f/c44_1 + 1.0f/c44_2 + 1.0f/c44_3 + 1.0f/c44_4),-1.0f);

            Tyz[index] += dt*Myz*(dVy_dz + dVz_dy);
        }

        if ((i > 3) && (i < nzz-4) && (j > 3) && (j < nxx-4) && (k > 3) && (k < nyy-4))
            P[index] = (Txx[index] + Tyy[index] + Tzz[index]) / 3.0f;
    }
}